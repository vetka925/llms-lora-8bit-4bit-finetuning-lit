#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ < 700 && __CUDA_ARCH__ > 600
// adapted from https://github.com/torch/cutorch/blob/master/lib/THC/THCAtomics.cuh
__device__ __forceinline__ void atomicAddHalf(__half* address, c10::Half val) {
    unsigned int *address_as_ui = reinterpret_cast<unsigned int *>(reinterpret_cast<char *>(address) - (reinterpret_cast<size_t>(address) & 2));
    unsigned int old = *address_as_ui;
    unsigned int assumed;

    do {
        assumed = old;
        unsigned short hsum = reinterpret_cast<size_t>(address) & 2 ? (old >> 16) : (old & 0xffff);
        hsum += val;
        old = reinterpret_cast<size_t>(address) & 2
                 ? (old & 0xffff) | (hsum << 16)
                 : (old & 0xffff0000) | hsum;
        old = atomicCAS(address_as_ui, assumed, old);

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
}
#endif
#endif

template <typename scalar_t>
__global__ void VecQuant2MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
	int batch,
	int vec_height, 	
    int height,
    int width
);

template <typename scalar_t>
__global__ void VecQuant3MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
	int batch,
	int vec_height, 	
    int height,
    int width
);

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
	int batch,
	int vec_height, 	
    int height,
    int width
);

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
	int batch,
	int vec_height, 	
    int height,
    int width
);

const int BLOCKWIDTH  = 256;
const int BLOCKHEIGHT2 =  16;
const int BLOCKHEIGHT3 =  24;
const int BLOCKHEIGHT4 =  32; 
const int BLOCKHEIGHT8 =  64;

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

void vecquant2matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT2 - 1) / BLOCKHEIGHT2,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant2matmul_cuda", ([&] {
      VecQuant2MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        batch, vec_height, height, width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant2MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
	int batch,
	int vec_height,
    int height,
    int width
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT2 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[b * vec_height + (h / BLOCKHEIGHT2) * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t scale = scales[w];
  scalar_t zero = zeros[w];

  scalar_t res = 0;
  int i = width * h + w;
  int k = 0;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);
    res += (scale * scalar_t((tmp >> 0) & 0x3) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp >> 2) & 0x3) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp >> 4) & 0x3) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp >> 6) & 0x3) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp >> 8) & 0x3) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp >> 10) & 0x3) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp >> 12) & 0x3) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp >> 14) & 0x3) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp >> 16) & 0x3) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp >> 18) & 0x3) - zero) * blockvec[k + 9];
    res += (scale * scalar_t((tmp >> 20) & 0x3) - zero) * blockvec[k + 10];
    res += (scale * scalar_t((tmp >> 22) & 0x3) - zero) * blockvec[k + 11];
    res += (scale * scalar_t((tmp >> 24) & 0x3) - zero) * blockvec[k + 12];
    res += (scale * scalar_t((tmp >> 26) & 0x3) - zero) * blockvec[k + 13];
    res += (scale * scalar_t((tmp >> 28) & 0x3) - zero) * blockvec[k + 14];
    res += (scale * scalar_t((tmp >> 30) & 0x3) - zero) * blockvec[k + 15];
    i += width;
    k += 16;
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant3matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT3 - 1) / BLOCKHEIGHT3,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant3matmul_cuda", ([&] {
      VecQuant3MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        batch, vec_height, height, width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant3MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
	int batch,
	int vec_height,
    int height,
    int width
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT3 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[b * vec_height + (h / BLOCKHEIGHT3) * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t scale = scales[w];
  scalar_t zero = zeros[w];

  scalar_t res = 0;
  int i = width * h + w;
  int k = 0;

  unsigned int tmp1;
  unsigned int tmp2;
  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp1 = as_unsigned(mat[i]);
    res += (scale * scalar_t((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
    i += width;
    tmp2 = as_unsigned(mat[i]);
    tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x4);
    tmp2 >>= 1;
    res += (scale * scalar_t(tmp) - zero) * blockvec[k + 10];
    k += 11;
    res += (scale * scalar_t((tmp2 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp2 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp2 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp2 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp2 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp2 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp2 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp2 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp2 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp2 >> 27) & 0x7) - zero) * blockvec[k + 9];
    i += width;
    tmp1 = as_unsigned(mat[i]);
    tmp = (tmp2 >> 30) | ((tmp1 << 1) & 0x6);
    tmp1 >>= 2;
    res += (scale * scalar_t(tmp) - zero) * blockvec[k + 10];
    k += 11;
    res += (scale * scalar_t((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
    i += width;
    k += 10;
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant4matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant4matmul_cuda", ([&] {
      VecQuant4MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        batch, vec_height, height, width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
	int batch,
	int vec_height,
    int height,
    int width
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT4 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[b * vec_height + (h / BLOCKHEIGHT4) * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t scale = scales[w];
  scalar_t zero = zeros[w];

  scalar_t res = 0;
  int i = width * h + w;
  int k = 0;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);
    res += (scale * scalar_t((tmp >> 0) & 0xF) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp >> 4) & 0xF) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp >> 8) & 0xF) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp >> 12) & 0xF) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp >> 16) & 0xF) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp >> 20) & 0xF) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp >> 24) & 0xF) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp >> 28) & 0xF) - zero) * blockvec[k + 7];
    i += width;
    k += 8;
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant8matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT8 - 1) / BLOCKHEIGHT8,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant8matmul_cuda", ([&] {
      VecQuant8MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        batch, vec_height, height, width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
	int batch,
	int vec_height,
    int height,
    int width
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT8 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[b * vec_height + (h / BLOCKHEIGHT8) * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t scale = scales[w];
  scalar_t zero = zeros[w];

  scalar_t res = 0;
  int i = width * h + w;
  int k = 0;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);
    res += (scale * scalar_t((tmp >> 0) & 0xFF) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp >> 8) & 0xFF) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp >> 16) & 0xFF) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp >> 24) & 0xFF) - zero) * blockvec[k + 3];
    i += width;
    k += 4;
  }

  atomicAdd(&mul[b * width + w], res);
}

template <typename scalar_t>
__global__ void VecQuant4TransposeMatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
	int batch,
	int vec_height,
    int height,
    int width
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT4 * blockIdx.x + threadIdx.x / 8;
  unsigned int shift = (unsigned int)((threadIdx.x % 8) * 4);
  int w = BLOCKWIDTH * blockIdx.y;
  
  int n_rows = 8 * BLOCKHEIGHT4 * blockIdx.x + threadIdx.x;
  int n_cols = b;
  
  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[n_cols * vec_height + w + threadIdx.x];
  __syncthreads();

  scalar_t res = 0;
  int i = width * h + w;
  int k = 0;
  int j = w;
  unsigned int tmp;
  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);
    res += (scales[j] * scalar_t((tmp >> shift) & 0xF) - zeros[j]) * blockvec[k];
    i += 1;
    j += 1;
    k += 1;
  }
  
  atomicAdd(&mul[n_cols * height * 8 + n_rows], res);
}

void vecquant4transposematmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  
  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant4transposematmul_cuda", ([&] {
      VecQuant4TransposeMatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        batch, vec_height, height, width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant4MatMulHalfKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
	int batch,
	int vec_height,
    int height,
    int width
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT4 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ __half blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = __half(vec[b * vec_height + (h / BLOCKHEIGHT4) * BLOCKWIDTH + threadIdx.x]);
  __syncthreads();

  __half scale = __half(scales[w]);
  __half zero = __half(zeros[w]);

  __half res = __float2half(0.0f);
  int i = width * h + w;
  int k = 0;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);
    res = __hadd(res, __hmul(__hsub(__hmul(scale, __int2half_rn((tmp >> 0) & 0xF)), zero), blockvec[k + 0]));
    res = __hadd(res, __hmul(__hsub(__hmul(scale, __int2half_rn((tmp >> 4) & 0xF)), zero), blockvec[k + 1]));
    res = __hadd(res, __hmul(__hsub(__hmul(scale, __int2half_rn((tmp >> 8) & 0xF)), zero), blockvec[k + 2]));
    res = __hadd(res, __hmul(__hsub(__hmul(scale, __int2half_rn((tmp >> 12) & 0xF)), zero), blockvec[k + 3]));
    res = __hadd(res, __hmul(__hsub(__hmul(scale, __int2half_rn((tmp >> 16) & 0xF)), zero), blockvec[k + 4]));
    res = __hadd(res, __hmul(__hsub(__hmul(scale, __int2half_rn((tmp >> 20) & 0xF)), zero), blockvec[k + 5]));
    res = __hadd(res, __hmul(__hsub(__hmul(scale, __int2half_rn((tmp >> 24) & 0xF)), zero), blockvec[k + 6]));
    res = __hadd(res, __hmul(__hsub(__hmul(scale, __int2half_rn((tmp >> 28) & 0xF)), zero), blockvec[k + 7]));
    i += width;
    k += 8;
  }
  
  __half* mul2 = (__half*)mul;
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ < 700 && __CUDA_ARCH__ > 600
  atomicAddHalf(&mul2[b * width + w], res);
#else
  atomicAdd(&mul2[b * width + w], res);
#endif
#endif

}

void vecquant4matmul_half_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_SWITCH(vec.type(), "vecquant4matmul_half_cuda",
    AT_DISPATCH_CASE(at::ScalarType::Half, ([&] {
      VecQuant4MatMulHalfKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        batch, vec_height, height, width
      );
    })
  ));
}

template <typename scalar_t>
__global__ void VecQuant4TransposeMatMulHalfKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
	int batch,
	int vec_height,
    int height,
    int width
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT4 * blockIdx.x + threadIdx.x / 8;
  unsigned int shift = (unsigned int)((threadIdx.x % 8) * 4);
  int w = BLOCKWIDTH * blockIdx.y;
  
  int n_rows = 8 * BLOCKHEIGHT4 * blockIdx.x + threadIdx.x;
  int n_cols = b;
  
  __shared__ __half blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = __half(vec[n_cols * vec_height + w + threadIdx.x]);
  __syncthreads();

  __half res = __float2half(0.0f);
  int i = width * h + w;
  int k = 0;
  int j = w;
  unsigned int tmp;
  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);
    __half zero = __half(zeros[j]);
    __half scale = __half(scales[j]);
    res = __hadd(res, __hmul(__hsub(__hmul(scale, __int2half_rn((tmp >> shift) & 0xF)), zero), blockvec[k]));
    i += 1;
    j += 1;
    k += 1;
  }
  
  __half* mul2 = (__half*)mul;
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ < 700 && __CUDA_ARCH__ > 600
  atomicAddHalf(&mul2[n_cols * height * 8 + n_rows], res);
#else
  atomicAdd(&mul2[n_cols * height * 8 + n_rows], res);
#endif
#endif
}

void vecquant4transposematmul_half_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  
  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_SWITCH(vec.type(), "vecquant4transposematmul_half_cuda",
    AT_DISPATCH_CASE(at::ScalarType::Half, ([&] {
      VecQuant4TransposeMatMulHalfKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        batch, vec_height, height, width
      );
    })
  ));
}

template <typename scalar_t>
__global__ void VecQuant4ReconsKernel(
    const       int* __restrict__ mat,
           scalar_t* __restrict__ res,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int height,
    int width
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT4 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;
  int n_rows = h * 8 + b;
  int n_cols = w;
  scalar_t scale = scales[w];
  scalar_t zero = zeros[w];
  int i = width * h + width * (b / 8) + w;
  int shift = b % 8 * 4;
  unsigned int tmp = as_unsigned(mat[i]);
  scalar_t result = (scale * scalar_t((tmp >> shift) & 0xF) - zero);
  res[n_rows * width + n_cols] = result;
}

void vecquant4recons_cuda(
  torch::Tensor mat,
  torch::Tensor res,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int batch = BLOCKWIDTH;
  int height = mat.size(0);
  int width = mat.size(1);
  
  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    scales.type(), "vecquant4recons_cuda", ([&] {
      VecQuant4ReconsKernel<<<blocks, threads>>>(
        mat.data<int>(), res.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<scalar_t>(),
        height, width
      );
    })
  );
}
